#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "mnist_file.h"
#include "neural_network.h"
#include "hyperparameters.h"
#include "cuda_kernels.h"

// Convert a pixel value from 0-255 to one from 0 to 1
#define PIXEL_SCALE(x) (((float) (x)) / 255.0f)

// Returns a random value between 0 and 1
#define RAND_FLOAT() (((float) rand()) / ((float) RAND_MAX))

/**
 * Initialise the weights and bias vectors with values between 0 and 1
 */
void neural_network_random_weights(neural_network_t * network) {
    int i, j;

    // First layer weights
    for (i = 0; i < HIDDEN_LAYER1_SIZE; i++) {
        network->b1[i] = RAND_FLOAT() - 0.5f;
        for (j = 0; j < INPUT_LAYER_SIZE; j++) {
            network->W1[i][j] = (RAND_FLOAT() - 0.5f) * 0.01f;
        }
    }

    // Second layer weights
    for (i = 0; i < HIDDEN_LAYER2_SIZE; i++) {
        network->b2[i] = RAND_FLOAT() - 0.5f;
        for (j = 0; j < HIDDEN_LAYER1_SIZE; j++) {
            network->W2[i][j] = (RAND_FLOAT() - 0.5f) * 0.01f;
        }
    }

    // Output layer weights
    for (i = 0; i < OUTPUT_LAYER_SIZE; i++) {
        network->b3[i] = RAND_FLOAT() - 0.5f;
        for (j = 0; j < HIDDEN_LAYER2_SIZE; j++) {
            network->W3[i][j] = (RAND_FLOAT() - 0.5f) * 0.01f;
        }
    }
}

/**
 * Use the weights and bias vector to forward propogate through the neural
 * network and calculate the activations for an input.
 */
void neural_network_hypothesis(mnist_image_t * image, neural_network_t * network, float activations[OUTPUT_LAYER_SIZE]) {
    float *d_W1, *d_W2, *d_W3, *d_b1, *d_b2, *d_b3;
    float *d_input, *d_layer1_activations, *d_layer2_activations, *d_output_activations;

    // Scale input pixels
    float scaled_pixels[INPUT_LAYER_SIZE];
    for (int i = 0; i < INPUT_LAYER_SIZE; i++) {
        scaled_pixels[i] = PIXEL_SCALE(image->pixels[i]);
    }

    // Allocate device memory
    hipMalloc(&d_W1, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_W2, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_W3, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b1, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_b2, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b3, OUTPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_input, INPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_layer1_activations, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_layer2_activations, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_output_activations, OUTPUT_LAYER_SIZE * sizeof(float));

    // Copy data to device
    hipMemcpy(d_W1, network->W1, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, network->W2, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W3, network->W3, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, network->b1, HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, network->b2, HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, network->b3, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input, scaled_pixels, INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernels
    dim3 blockDim(128);
    dim3 gridDim1((HIDDEN_LAYER1_SIZE + blockDim.x - 1) / blockDim.x);
    dim3 gridDim2((HIDDEN_LAYER2_SIZE + blockDim.x - 1) / blockDim.x);
    dim3 gridDim3((OUTPUT_LAYER_SIZE + blockDim.x - 1) / blockDim.x);

    matvec_mult<<<gridDim1, blockDim>>>(d_W1, d_input, d_b1, d_layer1_activations, HIDDEN_LAYER1_SIZE, INPUT_LAYER_SIZE);
    relu_activation<<<gridDim1, blockDim>>>(d_layer1_activations, HIDDEN_LAYER1_SIZE);

    matvec_mult<<<gridDim2, blockDim>>>(d_W2, d_layer1_activations, d_b2, d_layer2_activations, HIDDEN_LAYER2_SIZE, HIDDEN_LAYER1_SIZE);
    relu_activation<<<gridDim2, blockDim>>>(d_layer2_activations, HIDDEN_LAYER2_SIZE);

    matvec_mult<<<gridDim3, blockDim>>>(d_W3, d_layer2_activations, d_b3, d_output_activations, OUTPUT_LAYER_SIZE, HIDDEN_LAYER2_SIZE);
    softmax_activation<<<1, blockDim>>>(d_output_activations, d_output_activations, OUTPUT_LAYER_SIZE);

    // Copy results back to host
    hipMemcpy(activations, d_output_activations, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_W1);
    hipFree(d_W2);
    hipFree(d_W3);
    hipFree(d_b1);
    hipFree(d_b2);
    hipFree(d_b3);
    hipFree(d_input);
    hipFree(d_layer1_activations);
    hipFree(d_layer2_activations);
    hipFree(d_output_activations);
}

/**
 * Update the gradients for this step of gradient descent using the gradient
 * contributions from a single training example (image).
 * 
 * This function returns the loss contribution from this training example.
 */
float neural_network_gradient_update(mnist_image_t * image, neural_network_t * network, neural_network_gradient_t * gradient, uint8_t label) {
    float *d_W1, *d_W2, *d_W3, *d_b1, *d_b2, *d_b3;
    float *d_input, *d_layer1_activations, *d_layer2_activations, *d_output_activations;
    float *d_layer2_errors, *d_layer1_errors, *d_output_errors;
    float *d_W1_grad, *d_W2_grad, *d_W3_grad, *d_b1_grad, *d_b2_grad, *d_b3_grad;

    // Scale input pixels
    float scaled_pixels[INPUT_LAYER_SIZE];
    for (int i = 0; i < INPUT_LAYER_SIZE; i++) {
        scaled_pixels[i] = PIXEL_SCALE(image->pixels[i]);
    }

    // Allocate device memory
    hipMalloc(&d_W1, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_W2, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_W3, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b1, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_b2, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b3, OUTPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_input, INPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_layer1_activations, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_layer2_activations, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_output_activations, OUTPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_layer2_errors, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_layer1_errors, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_output_errors, OUTPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_W1_grad, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float));
    hipMalloc(&d_W2_grad, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_W3_grad, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b1_grad, HIDDEN_LAYER1_SIZE * sizeof(float));
    hipMalloc(&d_b2_grad, HIDDEN_LAYER2_SIZE * sizeof(float));
    hipMalloc(&d_b3_grad, OUTPUT_LAYER_SIZE * sizeof(float));

    // Copy data to device
    hipMemcpy(d_W1, network->W1, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, network->W2, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W3, network->W3, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, network->b1, HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, network->b2, HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, network->b3, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input, scaled_pixels, INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernels for forward pass
    dim3 blockDim(128);
    dim3 gridDim1((HIDDEN_LAYER1_SIZE + blockDim.x - 1) / blockDim.x);
    dim3 gridDim2((HIDDEN_LAYER2_SIZE + blockDim.x - 1) / blockDim.x);
    dim3 gridDim3((OUTPUT_LAYER_SIZE + blockDim.x - 1) / blockDim.x);

    matvec_mult<<<gridDim1, blockDim>>>(d_W1, d_input, d_b1, d_layer1_activations, HIDDEN_LAYER1_SIZE, INPUT_LAYER_SIZE);
    hipDeviceSynchronize();
    relu_activation<<<gridDim1, blockDim>>>(d_layer1_activations, HIDDEN_LAYER1_SIZE);
    hipDeviceSynchronize();

    matvec_mult<<<gridDim2, blockDim>>>(d_W2, d_layer1_activations, d_b2, d_layer2_activations, HIDDEN_LAYER2_SIZE, HIDDEN_LAYER1_SIZE);
    hipDeviceSynchronize();
    relu_activation<<<gridDim2, blockDim>>>(d_layer2_activations, HIDDEN_LAYER2_SIZE);
    hipDeviceSynchronize();

    matvec_mult<<<gridDim3, blockDim>>>(d_W3, d_layer2_activations, d_b3, d_output_activations, OUTPUT_LAYER_SIZE, HIDDEN_LAYER2_SIZE);
    hipDeviceSynchronize();
    softmax_activation<<<1, blockDim>>>(d_output_activations, d_output_activations, OUTPUT_LAYER_SIZE);
    hipDeviceSynchronize();

    // Copy output activations back to host
    hipMemcpy(output_activations, d_output_activations, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Backpropagation
    // Output layer error
    for (int i = 0; i < OUTPUT_LAYER_SIZE; i++) {
        output_errors[i] = (i == label) ? output_activations[i] - 1 : output_activations[i];
    }

    // Copy output errors to device
    hipMemcpy(d_output_errors, output_errors, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Backpropagation
    // Output layer error
    compute_output_layer_error<<<gridDim3, blockDim>>>(d_output_activations, d_output_errors, label, OUTPUT_LAYER_SIZE);
    hipDeviceSynchronize();

    // Hidden layer 2 error
    compute_hidden_layer_error<<<gridDim2, blockDim>>>(d_output_errors, d_W3, d_layer2_activations, d_layer2_errors, OUTPUT_LAYER_SIZE, HIDDEN_LAYER2_SIZE);
    hipDeviceSynchronize();

    // Hidden layer 1 error
    compute_hidden_layer_error<<<gridDim1, blockDim>>>(d_layer2_errors, d_W2, d_layer1_activations, d_layer1_errors, HIDDEN_LAYER2_SIZE, HIDDEN_LAYER1_SIZE);
    hipDeviceSynchronize();

    // Accumulate gradients for output layer
    accumulate_gradients<<<gridDim3, blockDim>>>(d_W3, d_b3, d_output_errors, d_layer2_activations, d_W3_grad, d_b3_grad, OUTPUT_LAYER_SIZE, HIDDEN_LAYER2_SIZE);
    hipDeviceSynchronize();

    // Accumulate gradients for hidden layer 2
    accumulate_gradients<<<gridDim2, blockDim>>>(d_W2, d_b2, d_layer2_errors, d_layer1_activations, d_W2_grad, d_b2_grad, HIDDEN_LAYER2_SIZE, HIDDEN_LAYER1_SIZE);
    hipDeviceSynchronize();

    // Accumulate gradients for hidden layer 1
    accumulate_gradients<<<gridDim1, blockDim>>>(d_W1, d_b1, d_layer1_errors, d_input, d_W1_grad, d_b1_grad, HIDDEN_LAYER1_SIZE, INPUT_LAYER_SIZE);
    hipDeviceSynchronize();

    // Copy gradients back to host
    hipMemcpy(gradient->W1_grad, d_W1_grad, HIDDEN_LAYER1_SIZE * INPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradient->W2_grad, d_W2_grad, HIDDEN_LAYER2_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradient->W3_grad, d_W3_grad, OUTPUT_LAYER_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradient->b1_grad, d_b1_grad, HIDDEN_LAYER1_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradient->b2_grad, d_b2_grad, HIDDEN_LAYER2_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradient->b3_grad, d_b3_grad, OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_W1);
    hipFree(d_W2);
    hipFree(d_W3);
    hipFree(d_b1);
    hipFree(d_b2);
    hipFree(d_b3);
    hipFree(d_input);
    hipFree(d_layer1_activations);
    hipFree(d_layer2_activations);
    hipFree(d_output_activations);
    hipFree(d_layer2_errors);
    hipFree(d_layer1_errors);
    hipFree(d_output_errors);
    hipFree(d_W1_grad);
    hipFree(d_W2_grad);
    hipFree(d_W3_grad);
    hipFree(d_b1_grad);
    hipFree(d_b2_grad);
    hipFree(d_b3_grad);

    // Cross-entropy loss for the output
    return 0.0f - log(output_activations[label]);
}

/**
 * Run one step of gradient descent and update the neural network. Return the total loss (sum of loss)
 */
float neural_network_training_step(mnist_dataset_t * dataset, neural_network_t * network, float learning_rate) {
    float total_loss = 0.0f;
    int i, j;

    // Initialize gradients to zero
    neural_network_gradient_t gradient;
    memset(&gradient, 0, sizeof(neural_network_gradient_t));

    /**
     * Calculate the Gradients and the Cross-Entropy Loss by looping through the training set.
     * The returned gradient is the sum of gradients from all inputs, not the average.
     */
    for (i = 0; i < dataset->size; i++) {
        total_loss += neural_network_gradient_update_gpu_v0(&dataset->images[i], network, &gradient, dataset->labels[i]);
    }

    // Update weights and biases
    for (i = 0; i < HIDDEN_LAYER1_SIZE; i++) {
        network->b1[i] -= learning_rate * gradient.b1_grad[i] / batch->size;
        for (j = 0; j < INPUT_LAYER_SIZE; j++) {
            network->W1[i][j] -= learning_rate * gradient.W1_grad[i][j] / batch->size;
        }
    }

    for (i = 0; i < HIDDEN_LAYER2_SIZE; i++) {
        network->b2[i] -= learning_rate * gradient.b2_grad[i] / batch->size;
        for (j = 0; j < HIDDEN_LAYER1_SIZE; j++) {
            network->W2[i][j] -= learning_rate * gradient.W2_grad[i][j] / batch->size;
        }
    }

    for (i = 0; i < OUTPUT_LAYER_SIZE; i++) {
        network->b3[i] -= learning_rate * gradient.b3_grad[i] / batch->size;
        for (j = 0; j < HIDDEN_LAYER2_SIZE; j++) {
            network->W3[i][j] -= learning_rate * gradient.W3_grad[i][j] / batch->size;
        }
    }
    
    return total_loss;
}

/**
 * Calculate the accuracy of the predictions of a neural network on a dataset.
 */
float calculate_accuracy(mnist_dataset_t * dataset, neural_network_t * network)
{
    float activations[MNIST_LABELS], max_activation;
    int i, j, correct, predict;

    // Loop through the dataset
    for (i = 0, correct = 0; i < dataset->size; i++) {
        // Calculate the activations for each image using the neural network
        neural_network_hypothesis(&dataset->images[i], network, activations);

        // Set predict to the index of the greatest activation
        for (j = 0, predict = 0, max_activation = activations[0]; j < MNIST_LABELS; j++) {
            if (max_activation < activations[j]) {
                max_activation = activations[j];
                predict = j;
            }
        }

        // Increment the correct count if we predicted the right label
        if (predict == dataset->labels[i]) {
            correct++;
        }
    }

    // Return the percentage we predicted correctly as the accuracy
    return ((float) correct) / ((float) dataset->size);
}

