/**
 *  Implements CUDA kernels for:
 *      Feedforward Computations:
 *          Matrix-vector multiplication.
 *          ReLU activation.
 *          Softmax activation.
 *      Backpropagation Computations:
 *          Error Calculation for Output Layer
 *          Gradient Computations (Matrix multiplications and reductions (summing gradients)
 *          Backpropagating the Error (Transpose and multiply operation)
 *          Weight and Bias Updates 
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <float.h>

// CUDA Kernel: Matrix-Vector Multiplication
__global__ void matvec_mult(float *W, float *x, float *b, float *out, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = b[row];
        for (int j = 0; j < cols; j++) {
            sum += W[row * cols + j] * x[j];
        }
        out[row] = sum;
    }
}

// CUDA Kernel: ReLU Activation
__global__ void relu_activation(float *input, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) input[i] = fmaxf(0.0f, input[i]);
}

// CUDA Kernel: Softmax Activation
__global__ void softmax_activation(float *input, float *output, int size) {
    __shared__ float shared_max[128];
    __shared__ float shared_sum[128];
    
    int tid = threadIdx.x;
    
    // Find max for numerical stability
    float local_max = -FLT_MAX;
    for (int i = tid; i < size; i += blockDim.x) {
        local_max = fmaxf(local_max, input[i]);
    }
    shared_max[tid] = local_max;
    __syncthreads();
    
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid + stride]);
        }
        __syncthreads();
    }
    float max_val = shared_max[0];
    __syncthreads();

    // Compute exp(x - max) for stability
    float local_sum = 0.0f;
    for (int i = tid; i < size; i += blockDim.x) {
        float val = expf(input[i] - max_val);
        output[i] = val;  // Store intermediate result
        local_sum += val;
    }
    shared_sum[tid] = local_sum;
    __syncthreads();
    
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();
    }
    float sum = shared_sum[0];
    
    // Normalize
    for (int i = tid; i < size; i += blockDim.x) {
        output[i] /= sum;
    }
}

__global__ void compute_output_layer_error(float *output_activations, float *output_errors, uint8_t label, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        output_errors[i] = (i == label) ? output_activations[i] - 1 : output_activations[i];
    }
}

__global__ void compute_hidden_layer_error(float *next_layer_errors, float *next_layer_weights, float *layer_activations, float *layer_errors, int next_layer_size, int layer_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < layer_size) {
        float error = 0.0f;
        for (int j = 0; j < next_layer_size; j++) {
            error += next_layer_errors[j] * next_layer_weights[j * layer_size + i];
        }
        layer_errors[i] = error * (layer_activations[i] > 0 ? 1.0f : 0.0f); // ReLU derivative
    }
}

__global__ void accumulate_gradients(float *weights, float *biases, float *errors, float *prev_activations, float *weights_grad, float *biases_grad, int layer_size, int prev_layer_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < layer_size) {
        atomicAdd(&biases_grad[i], errors[i]);
        for (int j = 0; j < prev_layer_size; j++) {
            atomicAdd(&weights_grad[i * prev_layer_size + j], errors[i] * prev_activations[j]);
        }
    }
}