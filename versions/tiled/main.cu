#include "hip/hip_runtime.h"
#include "tiled_kernel.h"
#include "neural_network.h"
#include "mnist_file.h"
#include "hyperparameters.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define INPUT_LAYER_SIZE      784       // MNIST-fashionfashion 28x28 image
#define HIDDEN_LAYER1_SIZE    128
#define HIDDEN_LAYER2_SIZE    128       // 2 hidden layers
#define OUTPUT_LAYER_SIZE     10        // MNIST-fashion 10 labels

#define BATCH_SIZE            32
#define LEARNING_RATE         0.5f
#define STEPS                 500    

// Global variable definitions
double activation_time = 0.0;
double relu_time = 0.0;
double softmax_time = 0.0;
double error_time = 0.0;
double gradient_time = 0.0;
double update_time = 0.0;
double training_time = 0.0;

int main() {
    // Initialize variables
    mnist_dataset_t *train_dataset, *test_dataset;
    mnist_dataset_t batch;
    neural_network_t network;
    /*network.W1 = (float*)malloc(INPUT_LAYER_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));  // Weights from input to first hidden layer
    network.b1 = (float*)malloc(HIDDEN_LAYER1_SIZE * sizeof(float));  // Biases for first hidden layer
    network.W2 = (float*)malloc(HIDDEN_LAYER1_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));  // Weights for second hidden layer
    network.b2 = (float*)malloc(HIDDEN_LAYER2_SIZE * sizeof(float));  // Biases for second hidden layer
    network.W3 = (float*)malloc(HIDDEN_LAYER2_SIZE * OUTPUT_LAYER_SIZE * sizeof(float));  // Weights from second hidden layer to output
    network.b3 = (float*)malloc(OUTPUT_LAYER_SIZE * sizeof(float));  // Biases for output layer*/
    // Dynamically allocate memory for weights and biases
    float* W1 = (float*)malloc(INPUT_LAYER_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));
    float* b1 = (float*)malloc(HIDDEN_LAYER1_SIZE * sizeof(float));

    float* W2 = (float*)malloc(HIDDEN_LAYER1_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));
    float* b2 = (float*)malloc(HIDDEN_LAYER2_SIZE * sizeof(float));

    float* W3 = (float*)malloc(HIDDEN_LAYER2_SIZE * OUTPUT_LAYER_SIZE * sizeof(float));
    float* b3 = (float*)malloc(OUTPUT_LAYER_SIZE * sizeof(float));

// Check if memory allocation was successful
    if (W1 == NULL || b1 == NULL || W2 == NULL || b2 == NULL || W3 == NULL || b3 == NULL) {
       printf("Memory allocation failed.\n");
       // Handle memory allocation failure
       return -1; // Exit the program or handle it as needed
    }

// Assign dynamically allocated memory to the network struct
    memcpy(network.W1, W1, INPUT_LAYER_SIZE * HIDDEN_LAYER1_SIZE * sizeof(float));
    memcpy(network.b1, b1, HIDDEN_LAYER1_SIZE * sizeof(float));

    memcpy(network.W2, W2, HIDDEN_LAYER1_SIZE * HIDDEN_LAYER2_SIZE * sizeof(float));
    memcpy(network.b2, b2, HIDDEN_LAYER2_SIZE * sizeof(float));

    memcpy(network.W3, W3, HIDDEN_LAYER2_SIZE * OUTPUT_LAYER_SIZE * sizeof(float));
    memcpy(network.b3, b3, OUTPUT_LAYER_SIZE * sizeof(float));

    // Load MNIST datasets
    train_dataset = mnist_get_dataset(TRAIN_IMAGES, TRAIN_LABELS);
    test_dataset = mnist_get_dataset(TEST_IMAGES, TEST_LABELS);

    // Initialise weights and biases with random values
    neural_network_random_weights(&network);

    // Allocate memory for CUDA
    // CUDA device memory for input, weights, and output
    float* d_input;
    float* d_W1;
    float* d_b1;
    float* d_W2;
    float* d_b2;
    float* d_W3;
    float* d_b3;
    float* d_output;
    int batches = train_dataset->size / BATCH_SIZE;

    hipMalloc(&d_input, BATCH_SIZE * 784 * sizeof(float));
    hipMalloc(&d_W1, 784 * 128 * sizeof(float));
    hipMalloc(&d_b1, 128 * sizeof(float));
    hipMalloc(&d_W2, 128 * 128 * sizeof(float));
    hipMalloc(&d_b2, 128 * sizeof(float));
    hipMalloc(&d_W3, 128 * 10 * sizeof(float));
    hipMalloc(&d_b3, 10 * sizeof(float));
    hipMalloc(&d_output, BATCH_SIZE * 10 * sizeof(float));

    hipMemcpy(d_W1, network.W1, 784 * 128 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, network.b1, 128 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, network.W2, 128 * 128 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, network.b2, 128 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W3, network.W3, 128 * 10 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, network.b3, 10 * sizeof(float), hipMemcpyHostToDevice);

    /*float* h_weights = (float*)malloc(784 * 10 * sizeof(float));
    for (int i = 0; i < 784 * 10; i++) {
        h_weights[i] = (float)(rand() % 10) / 10.0f;  // Random values between 0 and 1
    }*/

    // Copy weights to device
    //hipMemcpy(d_weights, h_weights, 784 * 10 * sizeof(float), hipMemcpyHostToDevice);

    // Training loop
    clock_t start, end;
    start = clock();

    //int batches = train_dataset->size / BATCH_SIZE;
    for (int i = 0; i < STEPS; i++) {
        // Initialise a new batch
        mnist_batch(train_dataset, &batch, BATCH_SIZE, i % batches);

        // Copy batch input to device
        //hipMemcpy(d_input, batch.images, BATCH_SIZE * 784 * sizeof(float), hipMemcpyHostToDevice);
        /*hipMemcpy(d_input, batch.images, BATCH_SIZE * 784 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W1, network.W1, 784 * 128 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b1, network.b1, 128 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W2, network.W2, 128 * 128 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b2, network.b2, 128 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W3, network.W3, 128 * 10 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b3, network.b3, 10 * sizeof(float), hipMemcpyHostToDevice);*/

        hipMemcpy(d_input, batch.images, BATCH_SIZE * 784 * sizeof(float), hipMemcpyHostToDevice);


        // Launch Tiled Shared Memory kernel for activation
        /*dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);*/
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((784 + BLOCK_SIZE - 1) / BLOCK_SIZE, (HIDDEN_LAYER1_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);



        /*hipEvent_t activation_start, activation_stop;
        hipEventCreate(&activation_start);
        hipEventCreate(&activation_stop);

        hipEventRecord(activation_start);
        tiledSharedMemoryKernel<<<grid, block>>>(d_input, d_weights, d_output, width, height);
        hipEventRecord(activation_stop);
        hipEventSynchronize(activation_stop);

        float activation_ms = 0;
        hipEventElapsedTime(&activation_ms, activation_start, activation_stop);
        activation_time += activation_ms / 1000.0;

        if (i % 25 == 0) {
            printf("Step %04d\tActivation Time: %.2f seconds\n", i, activation_time);
        }*/
        tiledSharedMemoryKernel<<<grid, block>>>(d_input, d_W1, d_output, 784, HIDDEN_LAYER1_SIZE);

        // Perform one step of training (forward pass, loss, backward pass, etc.)
        float total_loss = neural_network_training_step(&batch, &network, LEARNING_RATE);

        // Calculate accuracy
        float accuracy = calculate_accuracy(test_dataset, &network);

        // Print metrics every 25 steps
        if (i % 25 == 0) {
            printf("Step %04d\tAverage Loss: %.2f\tAccuracy: %.3f\n", i, total_loss / BATCH_SIZE, accuracy);
        }
    }

    end = clock();
    training_time = ((double)(end - start)) / CLOCKS_PER_SEC;

    printf("Training completed.\n\n");
    printf("Time for activation calculation: %f seconds\n", activation_time);
    printf("Time for ReLU calculation: %f seconds\n", relu_time);
    printf("Time for softmax calculation: %f seconds\n", softmax_time);
    printf("Time for error calculation: %f seconds\n", error_time);
    printf("Time for gradient calculation: %f seconds\n", gradient_time);
    printf("Time for weight update: %f seconds\n", update_time);
    printf("Total training time: %f seconds\n", training_time);

    // Cleanup
    mnist_free_dataset(train_dataset);
    mnist_free_dataset(test_dataset);
    /*hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);*/
    hipFree(d_input);
    hipFree(d_W1);
    hipFree(d_b1);
    hipFree(d_W2);
    hipFree(d_b2);
    hipFree(d_W3);
    hipFree(d_b3);
    hipFree(d_output);

    return 0;
}

